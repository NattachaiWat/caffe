#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/stn_pos_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void StnPosLossForwardGPU(const int N, const int cnt, const int L_pos, const Dtype threshold, 
    const Dtype* data, const int* pos, Dtype* loss_array)
{
    CUDA_KERNEL_LOOP(i, N) {
        Dtype mdist(0.0);
        for(int j=0; j<L_pos; j++)
        {
            mdist += threshold - data[i*cnt + pos[j]] > Dtype(0) ? threshold - data[i*cnt + pos[j]]: 0;
        }
        loss_array[i] = mdist;
    }
}

template <typename Dtype>
void StnPosLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    
    Dtype* loss_array = loss_.mutable_gpu_data();
    const Dtype* data = bottom[0]->mutable_gpu_data();
    const int* pos = pos_.gpu_data();
    int L_pos = pos_.count();
    int cnt = bottom[0]->count()/bottom[0]->num();
    
    StnPosLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, cnt, L_pos, 
        threshold, data, pos, loss_array);
    CUDA_POST_KERNEL_CHECK;

    Dtype loss = Dtype(0);
    caffe_gpu_asum(N, loss_.gpu_data(), &loss);
    loss /= N;
    top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void StnPosLossBackwardGPU(const int N, const int cnt, const int L_pos, const Dtype threshold,
    const Dtype* data, const int* pos, Dtype* diff)
{
    CUDA_KERNEL_LOOP(i, N) {
        Dtype mdist = Dtype(0);
        for(int j=0; j<L_pos; j++)
        {
            mdist = threshold - data[i*cnt + pos[j]] > Dtype(0) ? threshold - data[i*cnt + pos[j]] : 0;
            if (mdist > 0)
            {
                diff[i*cnt + pos[j]] = -1;
            }
        }
    }
}
template <typename Dtype>
void StnPosLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    
    const Dtype* data = bottom[0]->mutable_gpu_data();
    Dtype* diff = bottom[0]->mutable_gpu_diff();
    caffe_gpu_set(bottom[0]->count(), Dtype(0.), diff);
    const int* pos = pos_.gpu_data();
    int L_pos = pos_.count();
    int cnt = bottom[0]->count()/bottom[0]->num();
    
    StnPosLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, cnt, L_pos,
        threshold, data, pos, diff);
    CUDA_POST_KERNEL_CHECK;

    caffe_gpu_scale(bottom[0]->count(), top[0]->cpu_diff()[0]/N, diff, diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(StnPosLossLayer);
}

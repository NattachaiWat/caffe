#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/onlinepair_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include <cmath>
#include "caffe/util/other_functions.hpp"

namespace caffe {

template <typename Dtype>
void OnlinePairLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // const int count = bottom[0]->count();
  int num = bottom[0]->num();
  const int channels = bottom[0]->channels();
  Dtype margin = this->layer_param_.onlinepair_loss_param().margin();
  bool legacy_version =
      this->layer_param_.onlinepair_loss_param().legacy_version();
  int hards_pos = this->layer_param_.onlinepair_loss_param().hards_pos();
  int hards_neg = this->layer_param_.onlinepair_loss_param().hards_neg();
  // forward, step1, compute the distance matrix
  PairDist tmp;
  pairdist_neg_.clear();
  pairdist_pos_.clear();
  for (int i = 0; i<num-1; i++)
    for (int j = i+1; j < num; j++)
    {
        caffe_gpu_sub(
            channels, 
            bottom[0]->gpu_data() + (i*channels),
            bottom[0]->gpu_data() + (j*channels),
            diff_.mutable_gpu_data());
        Dtype dist_tmp;
        caffe_gpu_dot(channels, diff_.cpu_data(), diff_.cpu_data(), &(dist_tmp));
        tmp.dist = dist_tmp; 
        tmp.first = i;
        tmp.second = j;
        tmp.flag = 
            bottom[1]->mutable_cpu_data()[i] == bottom[1]->mutable_cpu_data()[j] ? 1 : 0;
        if (tmp.flag == 1)
        { pairdist_pos_.push_back( tmp ); }
        else
        { pairdist_neg_.push_back( tmp ); }
    }

  // sort the distance matrix
  std::sort( pairdist_pos_.begin(), pairdist_pos_.end(), pair_descend);
  std::sort( pairdist_neg_.begin(), pairdist_neg_.end(), pair_ascend);
  // sort_pairdist(pairdist_pos_, false);
  // sort_pairdist(pairdist_neg_, true);
  // take the first hards elements for backward and loss computation
  int pos_num = pairdist_pos_.size() > hards_pos ? hards_pos : pairdist_pos_.size();
  int neg_num = pairdist_neg_.size() > hards_neg ? hards_neg : pairdist_neg_.size();
  Dtype loss(0.0);
  for (int i = 0; i<pos_num; i++)
  {
    loss += pairdist_pos_[i].dist;
  }
  for (int i = 0; i<neg_num; i++)
  {
    if (legacy_version) 
    {
        loss += std::max( Dtype(0.0), margin - pairdist_neg_[i].dist);
    }
    else
    {
        Dtype dist = std::max( Dtype(0.0), margin - sqrt( pairdist_neg_[i].dist ) );
        loss += dist * dist;
    }
  }
  loss = loss / Dtype(pos_num + neg_num) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void OnlinePairLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Dtype margin = this->layer_param_.onlinepair_loss_param().margin();
  bool legacy_version =
    this->layer_param_.onlinepair_loss_param().legacy_version();
  int count = bottom[0]->count();
  const int channels = bottom[0]->channels();
  int hards_pos = this->layer_param_.onlinepair_loss_param().hards_pos();
  int hards_neg = this->layer_param_.onlinepair_loss_param().hards_neg();
  int pos_num = pairdist_pos_.size() > hards_pos ? hards_pos : pairdist_pos_.size();
  int neg_num = pairdist_neg_.size() > hards_neg ? hards_neg : pairdist_neg_.size();
  // the backward is here, first for similar and then for dissimilar pairs
  // before take all the gradient, reset it with zeros
  caffe_gpu_set(count, Dtype(0), bottom[0]->mutable_gpu_diff());
  int first;
  int second;
  // get the pointer of backward gradient
  Dtype* bout = bottom[0]->mutable_cpu_diff();
  std::cout<< "start the pos gradient computation" << std::endl;
  std::cout<< "pos count is : " << pos_num << std::endl;
  for (int i = 0; i < pos_num; i++)
  {
      first = pairdist_pos_[i].first;
      second = pairdist_pos_[i].second;
      // compute the diff 
      caffe_gpu_sub(
        channels,
        bottom[0]->gpu_data() + first*channels,
        bottom[0]->gpu_data() + second*channels,
        diff_.mutable_gpu_data());
      for (int j = 0; j < 2; j++)
      {
        Dtype sign = (j == 0) ? 1 : -1;
        int position = (j == 0)? first : second;
        Dtype alpha = sign * top[0]->cpu_diff()[0] / Dtype(pos_num + neg_num);
        // update the gradient
        caffe_gpu_axpby(
            channels,
            alpha,
            diff_.gpu_data(),
            Dtype(1.0),
            bout + (position*channels));
      }
  }
  std::cout<< "start the neg gradient computation" << std::endl;
  std::cout<< "neg count is : " << neg_num << std::endl;
  for (int i = 0; i < neg_num; i ++)
  {
    first = pairdist_neg_[i].first;
    second = pairdist_neg_[i].second;
    // compute the a-b, first - second
    caffe_gpu_sub(
        channels,
        bottom[0]->gpu_data() + first*channels,
        bottom[0]->gpu_data() + second*channels,
        diff_.mutable_gpu_data());
    for (int j = 0; j < 2; j++)
    {
        Dtype sign = (j == 0) ? 1 : -1;
        int position = (j == 0)? first : second;
        Dtype alpha = sign * top[0]->cpu_diff()[0] / Dtype(pos_num + neg_num);
        Dtype mdist(0.0);
        Dtype beta(0.0);
        if (legacy_version) {
            mdist = margin - pairdist_neg_[i].dist;
            beta = -alpha;
        }
        else {
            Dtype dist = sqrt(pairdist_neg_[i].dist);
            mdist = margin - dist;
            beta  = -alpha * mdist / (dist + Dtype(1e-4));
        }
        if ( mdist > Dtype(0.0) )
        {
            caffe_gpu_axpby(
                channels,
                beta,
                diff_.gpu_data(),
                Dtype(1.0),
                bout + (position*channels));
        }
    }
  }

}

INSTANTIATE_LAYER_GPU_FUNCS(OnlinePairLossLayer);

}  // namespace caffe

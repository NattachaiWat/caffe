#include "hip/hip_runtime.h"
/*
*
*=============================================
* for batch based global structure loss
*
* dangweili@gmail.com
*
*============================================
*/
#include <vector>

#include "caffe/layers/global_structure_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void GlobalStructureLossLayer<Dtype>::Forward_gpu( 
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
    // the forward function
    // create the sparse codes
    int num = bottom[1]->num();
    int count = bottom[0]->count();
    Dtype label(0);
    Dtype* data1 = NULL;
    Dtype* data2 = NULL;
    Dtype* data3 = NULL;
    data1 = sparse_codes_.mutable_cpu_data();
    data2 = bottom[1]->mutable_cpu_data();
    for(int i=0; i<num; i++)
    {   
        label = data2[i];
        data1[i + int(class_label[label][0])*num] = 1;
    }
    // compute the class mean value
    caffe_set(class_centers_.count(), Dtype(0), class_centers_.mutable_cpu_data());
    data1 = bottom[1]->mutable_cpu_data();
    for(int i=0; i<num; i++)
    {
        label = data1[i];
        data2 = bottom[0]->mutable_gpu_data() + i*D;
        data3 = class_centers_.mutable_gpu_data() + int(class_label[label][0])*D;
        caffe_gpu_add(D, data2, data3, data3);
    }
    typename map<Dtype, vector<Dtype> >::iterator it;
    for(it = class_label.begin(); it!=class_label.end(); it++)
    {   
        data1 = class_centers_.mutable_gpu_data() + int(it->second[0])*D;
        caffe_gpu_scale(D, Dtype(1.0/(it->second[1])), data1, data1);
    }
    // create the N*D center matrix
    data1 = sparse_codes_.mutable_gpu_data();
    data2 = class_centers_.mutable_gpu_data();
    data3 = center_matrix_.mutable_gpu_data();
    caffe_gpu_gemm(CblasTrans, CblasNoTrans, num, D, C, 
        Dtype(1), data1, data2, Dtype(0), data3); // need to be done
    // create the diff_xi_center_ matrix
    data1 = bottom[0]->mutable_gpu_data();
    data2 = center_matrix_.mutable_gpu_data();
    data3 = diff_xi_center_.mutable_gpu_data();
    caffe_gpu_sub(count, data1, data2, data3);
    // create the diff_centers_centers_
    data1 = class_centers_.mutable_gpu_data();
    data2 = extend_vector_.mutable_gpu_data();
    data3 = diff_centers_centers_.mutable_gpu_data();
    // copy the centers with C times, use each center to sub this matrix
    for (int i=0; i<C; i++)
    {
        // copy
        caffe_copy(C*D, data1, &data3[i*C*D]);
        // utilize the C to fast compute each center's difference with other centers
        caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, D, C, 1,
            Dtype(1), &data1[i*D], data2, Dtype(-1), &data3[i*C*D]);
    }
    // compute the loss
    float margin = this->layer_param_.global_structure_loss_param().margin();
    // record the loss in forward for each pair of centers
    data1 = delta_flag_.mutable_cpu_data(); 
    data2 = diff_centers_centers_.mutable_gpu_data();
    // could be optimized with half computation and then copy 
    Dtype loss_inter = 0;
    for (int i=0; i<C-1; i++)
    {
        for(int j=i+1; j<C; j++)
        {
            int offset = (i*C+j)*D;
            Dtype tmp = 0;
            caffe_gpu_dot(D, data2+offset, data2+offset, &tmp);
            Dtype mdist = std::max(Dtype(0), margin - tmp);
            loss_inter += mdist;
            data1[i*C + j] = mdist;
            data1[j*C + i] = mdist;
        }
    }
    if (C > 1)
        loss_inter = loss_inter*2/C/(C-1);
    else
        loss_inter = 0;
    // compute the intra loss
    Dtype loss_intra = 0;
    data1 = diff_xi_center_.mutable_gpu_data(); // the data that has been mean subtracted
    data2 = bottom[1]->mutable_cpu_data(); // the label 
    for(int i=0; i<num; i++)
    {
        int offset = i*D;
        Dtype tmp = 0;
        caffe_gpu_dot(D, data1+offset, data1+offset, &tmp);
        loss_intra += tmp/class_label[data2[i]][1];
    }
    loss_intra = loss_intra/2/C;
    Dtype weight = this->layer_param_.global_structure_loss_param().weight();
    Dtype loss = loss_intra + weight*loss_inter;
    top[0]->mutable_cpu_data()[0] = loss;
    top[0]->mutable_cpu_data()[1] = loss_intra;
    top[0]->mutable_cpu_data()[2] = loss_inter;
}

template <typename Dtype>
void GlobalStructureLossLayer<Dtype>::Backward_gpu( const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    // the backward function
    // using cached diff_xi_center_, diff_centers_centers_ and  delta_flag_ to update the diff
    Dtype weight = this->layer_param_.global_structure_loss_param().weight();
    Dtype loss_weight = top[0]->cpu_diff()[0];
    // update the diff from the intra loss
    int num = bottom[0]->num();
    Dtype* data1 = bottom[1]->mutable_cpu_data(); // label
    Dtype* data2 = diff_xi_center_.mutable_gpu_data(); // the intra diff
    Dtype* data3 = bottom[0]->mutable_gpu_diff(); // the diff data
    // reset the delta_flag_ with 0 and 1
    for(int i=0; i<C; i++)
        for(int j=0; j<C; j++)
            delta_flag_.mutable_cpu_data()[i*C+j] = delta_flag_.mutable_cpu_data()[i*C+j] > 0 ? 1 : 0;
    Dtype* data4 = delta_flag_.mutable_gpu_data();
    Dtype* data5 = diff_centers_centers_.mutable_gpu_data();
    // set the update
    for(int i=0; i<num; i++)
    {
        int label = int(class_label[data1[i]][0]);
        int offset = i*D;
        // update the diff from the intra loss
        caffe_copy(D, data2+offset, data3+offset);
        // update the diff from the inter loss
        Dtype scale = 0;
        if (C > 1)
            scale = weight*(-2)/(C-1);
        caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, 1, D, C,
            scale, data4+label*C, data5+label*C*D, Dtype(1.0), data3+offset);
        // scale the loss
        caffe_gpu_scale(D, Dtype(loss_weight/C/class_label[data1[i]][1]), data3+offset, data3+offset);
    }
}   

INSTANTIATE_LAYER_GPU_FUNCS(GlobalStructureLossLayer);
} // end namespace of caffe

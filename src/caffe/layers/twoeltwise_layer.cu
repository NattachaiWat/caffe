#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layers/twoeltwise_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaxForward(const int nthreads, const Dtype* bottom_data_a,
    const Dtype* bottom_data_b, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    if (bottom_data_a[index] > bottom_data_b[index]) {
        top_data[index] = bottom_data_a[index];
    } else {
      top_data[index] = bottom_data_b[index]; 
    }
  }
}

template <typename Dtype>
void TwoEltwiseLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  const int num = bottom[0]->num();
  const int chw = count/num;
  Dtype* top_data = preoutput_.mutable_gpu_data();
  Dtype* output_data = top[0]->mutable_gpu_data();
  
  switch (op_) {
  case TwoEltwiseParameter_TwoEltwiseOp_PROD:
    caffe_gpu_mul(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
        top_data);
    break;
  case TwoEltwiseParameter_TwoEltwiseOp_SUM:
    caffe_gpu_add(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
        top_data);
    break;
  case TwoEltwiseParameter_TwoEltwiseOp_MAX:
    MaxForward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom[0]->gpu_data(), bottom[1]->gpu_data(), top_data);
    break;
  case TwoEltwiseParameter_TwoEltwiseOp_SUB:
    caffe_gpu_sub(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
        top_data);
    break;
  default:
    LOG(FATAL) << "Unknown elementwise operation.";
  }
  if (this->layer_param_.twoeltwise_param().absout() == true )
  {
    caffe_gpu_abs(count, top_data, top_data);
  }
  if( this->layer_param_.twoeltwise_param().numsqsum() == true )
  {
    caffe_gpu_powx(count, preoutput_.mutable_gpu_data(), Dtype(2), preoutput_.mutable_gpu_data());
    std::cout << "li" << num << " " << chw << std::endl;
    caffe_gpu_gemv(CblasNoTrans, num, chw, Dtype(1.0), preoutput_.mutable_gpu_data(), 
            summer_vec_.mutable_gpu_data(), Dtype(0.0), top[0]->mutable_gpu_data());
  }else{
    caffe_copy(count, top_data, output_data);
  }
}

template <typename Dtype>
void TwoEltwiseLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{
    for (int i = 0; i < propagate_down.size(); ++i) {
      if (propagate_down[i]) { NOT_IMPLEMENTED; }
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(TwoEltwiseLayer);

}  // namespace caffe

#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <iostream>

#include "caffe/layers/randompair_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include <cmath>
#include "caffe/util/other_functions.hpp"
#include "caffe/util/rng.hpp"
using namespace std;

namespace caffe {

template <typename Dtype>
void RandomPairLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // const int count = bottom[0]->count();
  int num = bottom[0]->num();
  const int channels = bottom[0]->channels();
  Dtype margin_pos = this->layer_param_.randompair_loss_param().margin_pos();
  Dtype margin_neg = this->layer_param_.randompair_loss_param().margin_neg();
  bool legacy_version =
      this->layer_param_.randompair_loss_param().legacy_version();
  int hards_pos = this->layer_param_.randompair_loss_param().hards_pos();
  int hards_neg = this->layer_param_.randompair_loss_param().hards_neg();
  // forward, step1, compute the distance matrix
  PairDist tmp;
  pairdist_neg_.clear();
  pairdist_pos_.clear();
  for (int i = 0; i<num-1; i++)
    for (int j = i+1; j < num; j++)
    {
        tmp.first = i;
        tmp.second = j;
        tmp.flag = 
            bottom[1]->mutable_cpu_data()[i] == bottom[1]->mutable_cpu_data()[j] ? 1 : 0;
        if (tmp.flag == 1)
        { pairdist_pos_.push_back( tmp ); }
        else
        { pairdist_neg_.push_back( tmp ); }
    }
  // sort function
  caffe::rng_t* prefetch_rng =
      static_cast<caffe::rng_t*>(prefetch_rng_->generator());
  shuffle(pairdist_pos_.begin(), pairdist_pos_.end(), prefetch_rng);
  prefetch_rng =
      static_cast<caffe::rng_t*>(prefetch_rng_->generator());
  shuffle(pairdist_neg_.begin(), pairdist_neg_.end(), prefetch_rng);
  // sort the distance matrix
  // std::sort( pairdist_pos_.begin(), pairdist_pos_.end(), pair_descend);
  // std::sort( pairdist_neg_.begin(), pairdist_neg_.end(), pair_ascend);

  //if ( pairdist_pos_.size() > 0) 
  //{
  //  std::qsort(&pairdist_pos_[0], pairdist_pos_.size(), sizeof(PairDist), pair_descend_qsort);
  //}
  //if ( pairdist_neg_.size() > 0)
  //{
  //  std::qsort(&pairdist_neg_[0], pairdist_neg_.size(), sizeof(PairDist), pair_ascend_qsort); 
  //}


  // take the first hards elements for backward and loss computation
  int pos_num = pairdist_pos_.size() > hards_pos ? hards_pos : pairdist_pos_.size();
  int neg_num = pairdist_neg_.size() > hards_neg ? hards_neg : pairdist_neg_.size();
  
  for(int i=0; i<pos_num; i++)
  {
    caffe_gpu_sub(
        channels, 
        bottom[0]->gpu_data() + (pairdist_pos_[i].first*channels),
        bottom[0]->gpu_data() + (pairdist_pos_[i].second*channels),
        diff_.mutable_gpu_data());
    Dtype dist_tmp;
    caffe_gpu_dot(channels, diff_.cpu_data(), diff_.cpu_data(), &(dist_tmp));
    pairdist_pos_[i].dist = dist_tmp; 
  }
  for(int i=0; i<neg_num; i++)
  {
    caffe_gpu_sub(
        channels, 
        bottom[0]->gpu_data() + (pairdist_neg_[i].first*channels),
        bottom[0]->gpu_data() + (pairdist_neg_[i].second*channels),
        diff_.mutable_gpu_data());
    Dtype dist_tmp;
    caffe_gpu_dot(channels, diff_.cpu_data(), diff_.cpu_data(), &(dist_tmp));
    pairdist_neg_[i].dist = dist_tmp; 
  }

  Dtype loss(0.0);
  for (int i = 0; i<pos_num; i++)
  {
    if (legacy_version)
    {
        loss += std::max( Dtype(0.0), pairdist_pos_[i].dist - margin_pos);
    }
    else
    {
        Dtype dist = std::max( Dtype(0.0), sqrt( pairdist_pos_[i].dist)-margin_pos );
        loss += dist * dist;
    }
    // loss += pairdist_pos_[i].dist;
  }
  for (int i = 0; i<neg_num; i++)
  {
    if (legacy_version) 
    {
        loss += std::max( Dtype(0.0), margin_neg - pairdist_neg_[i].dist);
    }
    else
    {
        Dtype dist = std::max( Dtype(0.0), margin_neg - sqrt( pairdist_neg_[i].dist ) );
        loss += dist * dist;
    }
  }
  loss = loss / Dtype(pos_num + neg_num) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void RandomPairLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Dtype margin_pos = this->layer_param_.randompair_loss_param().margin_pos();
  Dtype margin_neg = this->layer_param_.randompair_loss_param().margin_neg();
  bool legacy_version =
    this->layer_param_.randompair_loss_param().legacy_version();
  int count = bottom[0]->count();
  const int channels = bottom[0]->channels();
  int hards_pos = this->layer_param_.randompair_loss_param().hards_pos();
  int hards_neg = this->layer_param_.randompair_loss_param().hards_neg();
  int pos_num = pairdist_pos_.size() > hards_pos ? hards_pos : pairdist_pos_.size();
  int neg_num = pairdist_neg_.size() > hards_neg ? hards_neg : pairdist_neg_.size();
  // the backward is here, first for similar and then for dissimilar pairs
  // before take all the gradient, reset it with zeros
  caffe_gpu_set(count, Dtype(0), bottom[0]->mutable_gpu_diff());
  int first;
  int second;
  // get the pointer of backward gradient
  Dtype* bout = bottom[0]->mutable_cpu_diff();
  // std::cout<< "start the pos gradient computation" << std::endl;
  // std::cout<< "pos count is : " << pos_num << std::endl;
  for (int i = 0; i < pos_num; i++)
  {
      first = pairdist_pos_[i].first;
      second = pairdist_pos_[i].second;
      // compute the diff 
      caffe_gpu_sub(
        channels,
        bottom[0]->gpu_data() + first*channels,
        bottom[0]->gpu_data() + second*channels,
        diff_.mutable_gpu_data());
      for (int j = 0; j < 2; j++)
      {
        Dtype sign = (j == 0) ? 1 : -1;
        int position = (j == 0)? first : second;
        Dtype alpha = sign * top[0]->cpu_diff()[0] / Dtype(pos_num + neg_num);
        Dtype mdist(0.0);
        Dtype beta(0.0);
        if (legacy_version)
        {
            mdist = pairdist_pos_[i].dist - margin_pos;
            beta = alpha;
        }
        else
        {
            Dtype dist = sqrt(pairdist_pos_[i].dist);
            mdist = dist - margin_pos;
            beta = alpha * mdist / (dist + Dtype(1e-4)); 
        }
        if (mdist > Dtype(0.0))
        {
            caffe_gpu_axpby(
                channels,
                beta,
                diff_.gpu_data(),
                Dtype(1.0),
                bout + (position*channels));
        }
      }
  }
  // std::cout<< "start the neg gradient computation" << std::endl;
  // std::cout<< "neg count is : " << neg_num << std::endl;
  for (int i = 0; i < neg_num; i ++)
  {
    first = pairdist_neg_[i].first;
    second = pairdist_neg_[i].second;
    // compute the a-b, first - second
    caffe_gpu_sub(
        channels,
        bottom[0]->gpu_data() + first*channels,
        bottom[0]->gpu_data() + second*channels,
        diff_.mutable_gpu_data());
    for (int j = 0; j < 2; j++)
    {
        Dtype sign = (j == 0) ? 1 : -1;
        int position = (j == 0)? first : second;
        Dtype alpha = sign * top[0]->cpu_diff()[0] / Dtype(pos_num + neg_num);
        Dtype mdist(0.0);
        Dtype beta(0.0);
        if (legacy_version) {
            mdist = margin_neg - pairdist_neg_[i].dist;
            beta = -alpha;
        }
        else {
            Dtype dist = sqrt(pairdist_neg_[i].dist);
            mdist = margin_neg - dist;
            beta  = -alpha * mdist / (dist + Dtype(1e-4));
        }
        if ( mdist > Dtype(0.0) )
        {
            caffe_gpu_axpby(
                channels,
                beta,
                diff_.gpu_data(),
                Dtype(1.0),
                bout + (position*channels));
        }
    }
  }

}

INSTANTIATE_LAYER_GPU_FUNCS(RandomPairLossLayer);

}  // namespace caffe

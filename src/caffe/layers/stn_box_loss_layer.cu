#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/stn_box_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void StnBoxLossForwardGPU(const int N, const Dtype threshold, 
    const Dtype* data, Dtype* loss_array)
{
    CUDA_KERNEL_LOOP(i, N) {
        Dtype mdist = Dtype(0);
        Dtype temp = Dtype(0);
        temp = (data[i*4+1]-data[i*4])*(data[i*4+1]-data[i*4]) - threshold;
        mdist += temp > 0 ? temp : 0;
        temp = (data[i*4+1]+data[i*4])*(data[i*4+1]+data[i*4]) - threshold;
        mdist += temp > 0 ? temp : 0;
        temp = (data[i*4+3]-data[i*4+2])*(data[i*4+3]-data[i*4+2]) - threshold;
        mdist += temp > 0 ? temp : 0;
        temp = (data[i*4+3]+data[i*4+2])*(data[i*4+3]+data[i*4+2]) - threshold;
        mdist += temp > 0 ? temp : 0;
        loss_array[i] = mdist/2;
    }
}

template <typename Dtype>
void StnBoxLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    
    Dtype* data = bottom[0]->mutable_gpu_data();
    Dtype* loss_array = loss_.mutable_gpu_data();

    StnBoxLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, threshold,
        data, loss_array);
    CUDA_POST_KERNEL_CHECK;

    Dtype loss;
    caffe_gpu_asum(N, loss_array, &loss);
    loss /= N;
    
    top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void StnBoxLossBackwardGPU(const int N, const Dtype threshold,  
    const Dtype* data, Dtype* diff)
{
    CUDA_KERNEL_LOOP(i, N) {
        Dtype mdist = Dtype(0);
        mdist = (data[i*4+1]-data[i*4])*(data[i*4+1]-data[i*4]) - threshold;
        if (mdist > 0)
        {
            diff[i*4+1] += data[i*4+1]-data[i*4];
            diff[i*4] += -1*(data[i*4+1]-data[i*4]);
        }
        mdist = (data[i*4+1]+data[i*4])*(data[i*4+1]+data[i*4]) - threshold;
        if (mdist > 0)
        {
            diff[i*4+1] += data[i*4+1]+data[i*4];
            diff[i*4] += data[i*4+1]+data[i*4];
        }
        mdist = (data[i*4+3]-data[i*4+2])*(data[i*4+3]-data[i*4+2]) - threshold;
        if (mdist > 0)
        {
            diff[i*4+3] += data[i*4+3]-data[i*4+2];
            diff[i*4+2] += -1*(data[i*4+3]-data[i*4+2]);
        }
        mdist = (data[i*4+3]+data[i*4+2])*(data[i*4+3]+data[i*4+2]) - threshold;
        if (mdist > 0)
        {
            diff[i*4+3] += data[i*4+3]+data[i*4+2];
            diff[i*4+2] += data[i*4+3]+data[i*4+2];
        }
    }
}
template <typename Dtype>
void StnBoxLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    
    Dtype* data = bottom[0]->mutable_gpu_data();
    Dtype* diff = bottom[0]->mutable_gpu_diff();
    caffe_gpu_set(bottom[0]->count(), Dtype(0), diff);

    StnBoxLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, threshold,
        data, diff); 
    CUDA_POST_KERNEL_CHECK;

    caffe_gpu_scale(bottom[0]->count(), top[0]->cpu_diff()[0]/N, diff, diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(StnBoxLossLayer);
}

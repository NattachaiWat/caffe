/*
* triplet_loss_layer.cu
*
*/

#include <algorithm>
#include <vector>


#include "caffe/layers/triplet_accuracy_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void TripletAccuracyLayer<Dtype>::Forward_gpu(
   const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
   const int count = bottom[0]->count();
   caffe_gpu_sub(
     count,
     bottom[0]->gpu_data(),  // a
     bottom[1]->gpu_data(),  // p
     diff_ap_.mutable_gpu_data());  // a_i-p_i
   caffe_gpu_mul(
     count,
     diff_ap_.mutable_gpu_data(),  // a_i-p_i
     diff_ap_.mutable_gpu_data(),
     diff_ap_.mutable_gpu_data());  // (a_i-p_i)^2
   caffe_gpu_sub(
     count,
     bottom[0]->gpu_data(),  // a
     bottom[2]->gpu_data(),  // n
     diff_an_.mutable_gpu_data());  // a_i-n_i
   caffe_gpu_mul(
     count,
     diff_an_.mutable_gpu_data(),  // a_i-n_i
     diff_an_.mutable_gpu_data(),
     diff_an_.mutable_gpu_data());  // (a_i-n_i)^2
   caffe_gpu_gemv(
     CblasNoTrans,
     bottom[0]->num(),
     bottom[0]->channels(),
     Dtype(1.0),                                         //alpha
     diff_ap_.gpu_data(),  // (a_i-p_i)^2                // A
     summer_vec_channel_.gpu_data(),                             // x
     Dtype(0.0),                                         //belta
     dist_sq_ap_.mutable_gpu_data());  // \Sum (a_i-p_i)^2  //y
   caffe_gpu_gemv(
     CblasNoTrans,
     bottom[0]->num(),
     bottom[0]->channels(),
     Dtype(1.0),                                         //alpha
     diff_an_.gpu_data(),  // (a_i-n_i)^2                // A
     summer_vec_channel_.gpu_data(),                             // x
     Dtype(0.0),                                         //belta
     dist_sq_an_.mutable_gpu_data());  // \Sum (a_i-n_i)^2  //y

   Dtype margin = this->layer_param_.triplet_accuracy_param().margin();
   Dtype accuracy = Dtype(0);
   for (int i = 0; i < bottom[0]->num(); ++i) {
     // std::cout<< "gpu diff is :" << margin << " "<< dist_sq_ap_.cpu_data()[i]<<" " <<dist_sq_an_.cpu_data()[i] << std::endl;
     accuracy += margin + dist_sq_ap_.cpu_data()[i] - dist_sq_an_.cpu_data()[i] >= 0 ? 0 : 1;
   }
   top[0]->mutable_cpu_data()[0] = accuracy/bottom[0]->num();
 }

template <typename Dtype>
void TripletAccuracyLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{
    for (int i = 0; i < propagate_down.size(); ++i) {
      if (propagate_down[i]) { NOT_IMPLEMENTED; }
    }
}
INSTANTIATE_LAYER_GPU_FUNCS(TripletAccuracyLayer);

}  // namespace caffe

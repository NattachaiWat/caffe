#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/stn_key_point_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void StnKeyPointLossForwardGPU(const int N, const int channels, const Dtype threshold, 
    const Dtype* data, const Dtype* gt_bias, const Dtype rate_hw, const int* gt_pos, Dtype* loss_array)
{
    CUDA_KERNEL_LOOP(i, N) {
        Dtype mdist(0.0);
        mdist = (data[i*channels + gt_pos[0]] - gt_bias[2*i+0])*(data[i*channels + gt_pos[0]] - gt_bias[2*i+0]);
        mdist += rate_hw*rate_hw*(data[i*channels + gt_pos[1]] - gt_bias[2*i+1])*(data[i*channels + gt_pos[1]] - gt_bias[2*i+1]);
        mdist = mdist - threshold;
        if (mdist > 0.0)
        {
            loss_array[i] = mdist/2;
        }
        else
        {
            loss_array[i] = 0.0;
        }
    }
}

template <typename Dtype>
void StnKeyPointLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    
    const Dtype* data = bottom[0]->gpu_data();
    Dtype* loss_array = loss_.mutable_gpu_data();
    caffe_gpu_set(loss_.count(), (Dtype)0, loss_array);

    const int channels = bottom[0]->channels();
    const Dtype* gt_bias = bottom[1]->gpu_data();
    const int* gt_pos = pos_.gpu_data();

    StnKeyPointLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(N_), CAFFE_CUDA_NUM_THREADS>>>(N_, channels, threshold_,
        data, gt_bias, rate_hw_, gt_pos, loss_array);
    CUDA_POST_KERNEL_CHECK; 

    Dtype loss;
    caffe_gpu_asum(N_, loss_array, &loss);
    loss /= N_;
    
    top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void StnKeyPointLossBackwardGPU(const int N, const int channels, 
    const Dtype* data, const Dtype* gt_bias, const int* gt_pos, const Dtype rate_hw, const Dtype* loss, Dtype* diff)
{
    CUDA_KERNEL_LOOP(i, N) {
        if (loss[i] > 0 )
        {
            int index = i*channels + gt_pos[0];
            diff[ index ] = data[ index ] - gt_bias[2*i+0];
            index = i*channels + gt_pos[1];
            diff[ index ] = rate_hw*(data[ index ] - gt_bias[2*i+1]);
        }
    }
}
template <typename Dtype>
void StnKeyPointLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    const Dtype* theta = bottom[0]->gpu_data();
    const Dtype* loss_array = loss_.gpu_data();
    const int channels = bottom[0]->channels();
    const Dtype* gt_bias = bottom[1]->gpu_data(); 
    const int* gt_pos = pos_.gpu_data();
    StnKeyPointLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(N_), CAFFE_CUDA_NUM_THREADS>>>(N_, channels, theta,
        gt_bias, gt_pos, rate_hw_, loss_array, bottom[0]->mutable_gpu_diff() ); 
    CUDA_POST_KERNEL_CHECK;

    caffe_gpu_scal(bottom[0]->count(), top[0]->cpu_diff()[0]/N_, bottom[0]->mutable_gpu_diff());
}
INSTANTIATE_LAYER_GPU_FUNCS(StnKeyPointLossLayer);
}
